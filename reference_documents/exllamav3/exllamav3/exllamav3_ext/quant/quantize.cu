#include "hip/hip_runtime.h"
#include "quantize.cuh"
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_fp16.h>
#include "../util.h"
#include "../util.cuh"
#include "codebook.cuh"
#include <cmath>

#define NUM_THREADS 1024

template <int K>
__global__ __launch_bounds__(1024)
void quantize_tiles_kernel
(
    const float* __restrict__ input_tiles_ptr,
    float* __restrict__ output_tiles_ptr,
    uint16_t* __restrict__ output_indices_ptr,
    float* __restrict__ temp_costs_ptr,
    uint16_t* __restrict__ temp_edges_ptr
)
{
    int tile_idx = blockIdx.x;

    constexpr int Kr = 16 - K;
    constexpr int max_q = 1 << K;
    constexpr int edges = 65536 >> K;

    const float* input_tile = input_tiles_ptr + 256 * tile_idx;
    float* output_tile = output_tiles_ptr + 256 * tile_idx;
    uint16_t* output_indices = output_indices_ptr + 256 * tile_idx;
    float* temp_costs = temp_costs_ptr + 2 * edges * tile_idx;
    float* temp_costs_inc = temp_costs + edges;
    uint16_t* temp_edges = temp_edges_ptr + 256 * edges * tile_idx;

    auto forward = [&](int roll, int pre_state)
    {
        // Each thread iterates over all weights in the tile
        for (int i = 0; i < 256; ++i)
        {
            int ri = (i + roll) % 256;

            // Swap buffers.
            // temp_costs_inc[z] is the cost/cumulative error of an incoming edge from state (z & edge_mask)
            float* t = temp_costs;
            temp_costs = temp_costs_inc;
            temp_costs_inc = t;

            for (int out_edge_idx = threadIdx.x; out_edge_idx < edges; out_edge_idx += NUM_THREADS)
            {
                float w = input_tile[ri];

                float min_err = INFINITY;
                int min_in_edge = 0;

                #pragma unroll
                for (int k = 0; k < max_q; ++k)
                {
                    int state = (k << Kr) | out_edge_idx;

                    float err = decode_pcb_f_diff(state, w);
                    err = err * err;

                    int in_edge_idx = state >> K;
                    if (i > 0)
                        err += temp_costs_inc[in_edge_idx];
                    else if (pre_state >= 0 && in_edge_idx != pre_state)
                        err = 1e30f;

                    if (err < min_err)
                    {
                        min_err = err;
                        min_in_edge = in_edge_idx;
                    }
                }

                temp_costs[out_edge_idx] = min_err;
                temp_edges[edges * ri + out_edge_idx] = (uint16_t) min_in_edge;
            }

            // Next iteration depends on costs computed by current iteration
            __syncthreads();
        }
    };

    auto argmin_cost = [&]()
    {
        // Find the final state with the lowest total cost. Return value is only valid in thread 0

        float local_min = 1e30f;
        int local_idx = -1;
        for (int e = threadIdx.x; e < edges; e += NUM_THREADS)
        {
            float v = temp_costs_inc[e];
            if (v < local_min)
            {
                local_min = v;
                local_idx = e;
            }
        }

        // Shuffle reduction
        int lane_id = threadIdx.x % 32;
        int warp_id = threadIdx.x / 32;

        #pragma unroll
        for (int offset = 16; offset > 0; offset >>= 1)
        {
            float other_min = __shfl_down_sync(0xffffffff, local_min, offset, 32);
            int other_idx = __shfl_down_sync(0xffffffff, local_idx, offset, 32);
            if (other_min < local_min)
            {
                local_min = other_min;
                local_idx = other_idx;
            }
        }

        __shared__ float s_min[32];
        __shared__ int s_idx[32];

        s_min[warp_id] = local_min;
        s_idx[warp_id] = local_idx;
        __syncthreads();

        if (warp_id == 0)
        {
            local_min = lane_id * 32 < edges ? s_min[lane_id] : 1e31f;
            local_idx = s_idx[lane_id];

            #pragma unroll
            for (int offset = 16; offset > 0; offset >>= 1)
            {
                float other_min = __shfl_down_sync(0xffffffff, local_min, offset, 32);
                int other_idx = __shfl_down_sync(0xffffffff, local_idx, offset, 32);
                if (other_min < local_min)
                {
                    local_min = other_min;
                    local_idx = other_idx;
                }
            }
        }

        return local_idx;
    };

    auto backward = [&](int roll, bool write, int edge)
    {
        // Construct output tile. Since the graph has to be walked, this will run in a single thread per block.
        // Profiling says this is not a bottleneck

        if (threadIdx.x == 0)
        {
            for (int i = 255; i >= 0; --i)
            {
                int ri = (i + roll) % 256;

                int prev_edge = (int) temp_edges[edges * ri + edge];
                int encoded = (prev_edge << K) | edge;
                edge = prev_edge;

                if (write)
                {
                    output_indices[ri] = (uint16_t) encoded;
                    output_tile[ri] = __half2float(decode_pcb(encoded));
                }
                else if (ri == 0) break;
            }
        }

        // Broadcast to block
        __shared__ int broadcast;
        if (threadIdx.x == 0) broadcast = edge;
        __syncthreads();
        edge = broadcast;

        return edge;
    };

    // Solve starting at position 128 find initial state for second pass
    forward(128, -1);
    int end_state = argmin_cost();
    end_state = backward(128, false, end_state);

    // Solve again from position 0 with tail-biting constraint
    forward(0, end_state);
    backward(0, true, end_state);
}

#define __(i) quantize_tiles_kernel<i>
constexpr auto quantize_tiles_kernel_instances = std::array
{
    __(1), __(2), __(3), __(4), __(5), __(6), __(7), __(8)
};
#undef __

/*
Quantize batch of tiles

input_tiles: shape (n, 256), float
output_tiles: shape (n, 256), float
output_indices: shape (n, 256), uint16_t (unpacked)
temp_costs: shape (max_bsz, 2, 65536 >> K), float (scratch space for Viterbi algorithm)
temp_edges: shape (max_bsz, 256, 65536 >> K), uint16_t (scratch space for Viterbi algorithm)
K: number of bits per weight (1..8)
*/

void quantize_tiles
(
    at::Tensor input_tiles,
    at::Tensor output_tiles,
    at::Tensor output_indices,
    at::Tensor temp_costs,
    at::Tensor temp_edges,
    int K
)
{
    const at::cuda::OptionalCUDAGuard device_guard(input_tiles.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    TORCH_CHECK_DIM(input_tiles, 2);
    TORCH_CHECK_SIZE(input_tiles, 1, 256);
    TORCH_CHECK_SHAPES_FULL(input_tiles, output_indices);
    TORCH_CHECK_DTYPE(input_tiles, kFloat);
    TORCH_CHECK_DTYPE(output_tiles, kFloat);
    TORCH_CHECK_DTYPE(output_indices, kShort);

    int edges = 65536 >> K;
    int threads = MIN(NUM_THREADS, edges);

    int num_tiles = input_tiles.size(0);

    TORCH_CHECK_DTYPE(temp_costs, kFloat);
    TORCH_CHECK_DIM(temp_costs, 3);
    TORCH_CHECK_SIZE(temp_costs, 1, 2);
    TORCH_CHECK_SIZE(temp_costs, 2, edges);

    TORCH_CHECK_DTYPE(temp_edges, kShort);
    TORCH_CHECK_DIM(temp_edges, 3);
    TORCH_CHECK_SIZE(temp_edges, 1, 256);
    TORCH_CHECK_SIZE(temp_edges, 2, edges);

    int max_batch_size = temp_costs.size(0);

    int batch_i = 0;
    do
    {
        int batch_j = MIN(batch_i + max_batch_size, num_tiles);

        const float* input_tiles_ptr = ((const float*) input_tiles.data_ptr()) + 256 * batch_i;
        float* output_tiles_ptr = ((float*) output_tiles.data_ptr()) + 256 * batch_i;
        uint16_t* output_indices_ptr = ((uint16_t*) output_indices.data_ptr()) + 256 * batch_i;
        float* temp_costs_ptr = (float*) temp_costs.data_ptr();
        uint16_t* temp_edges_ptr = (uint16_t*) temp_edges.data_ptr();

        int bsz = batch_j - batch_i;

        quantize_tiles_kernel_instances[K - 1]<<<bsz, threads, 0, stream>>>
        (
            input_tiles_ptr,
            output_tiles_ptr,
            output_indices_ptr,
            temp_costs_ptr,
            temp_edges_ptr
        );
        cuda_check(hipPeekAtLastError());

        batch_i = batch_j;
    }
    while (batch_i < num_tiles);
}

template <typename T>
__global__ //__launch_bounds__(64)
void decode_kernel
(
    const uint16_t* __restrict__ input_tiles_ptr,
    T* __restrict__ output_tiles_ptr,
    int cols
)
{
    int col = threadIdx.x + blockIdx.x * 64;
    if (col >= cols) return;
    int row = blockIdx.y;
    int idx = row * cols + col;

    uint16_t enc = input_tiles_ptr[idx];
    if constexpr (std::is_same_v<T, float>)
        output_tiles_ptr[idx] = __half2float(decode_pcb((uint64_t) enc));
    else
        output_tiles_ptr[idx] = decode_pcb((uint64_t) enc);
}

/*
Decode tensor

input_indices: uint16_t
output_tiles: float or half
*/

void decode
(
    at::Tensor input_indices,
    at::Tensor output_tiles
)
{
    const at::cuda::OptionalCUDAGuard device_guard(input_indices.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    TORCH_CHECK_DIM(input_indices, 2);
    TORCH_CHECK_SHAPES_FULL(input_indices, output_tiles);
    TORCH_CHECK_DTYPE(input_indices, kShort);

    int rows = input_indices.size(0);
    int cols = input_indices.size(1);

    dim3 blockDim(64);
    dim3 gridDim(cols / 64, rows);

    if (output_tiles.dtype() == at::kFloat)
        decode_kernel<<<gridDim, blockDim, 0, stream>>>
        (
            (const uint16_t*) input_indices.data_ptr(),
            (float*) output_tiles.data_ptr(),
            cols
        );
    else if (output_tiles.dtype() == at::kHalf)
        decode_kernel<<<gridDim, blockDim, 0, stream>>>
        (
            (const uint16_t*) input_indices.data_ptr(),
            (half*) output_tiles.data_ptr(),
            cols
        );
}


#define NUM_THREADS_TD 1024
#define MAX_BINS 1024

__global__ __launch_bounds__(NUM_THREADS_TD)
void test_distribution_kernel
(
    const float* __restrict__ input_ptr,
    float* __restrict__ dist_output_ptr,
    float* __restrict__ ref_output_ptr,
    uint64_t numel,
    uint64_t num_bins,
    float min_value,
    float max_value
)
{
    __shared__ int histogram[MAX_BINS];
    auto reset_histogram = [&]()
    {
        for (int i = threadIdx.x; i < num_bins; i += NUM_THREADS_TD)
            histogram[i] = 0;
        __syncthreads();
    };

    auto write_histogram = [&](float* output_ptr, uint64_t sc)
    {
        float scf = (float) sc;
        for (int i = threadIdx.x; i < num_bins; i += NUM_THREADS_TD)
            output_ptr[i] = ((float) histogram[i]) / scf;
        __syncthreads();
    };

    auto count = [&](float val)
    {
        val -= min_value;
        val /= (max_value - min_value);
        val *= (float) num_bins;
        int idx = (int) val;
        if (idx < 0) idx = 0;
        if (idx > num_bins - 1) idx = num_bins - 1;
        atomicAdd(&histogram[idx], 1);
    };

    reset_histogram();
    for (uint64_t i = threadIdx.x; i < 65536; i += NUM_THREADS_TD)
        count(decode_3inst_f((uint16_t) (i & 0xffff)));
    __syncthreads();
    write_histogram(ref_output_ptr, 65536);

    reset_histogram();
    for (uint64_t i = threadIdx.x; i < numel; i += NUM_THREADS_TD)
        count(input_ptr[i]);
    __syncthreads();
    write_histogram(dist_output_ptr, numel);
}

/*
Compare tensor distribution to codebook (not optimized)

input: tensor, float, any shape
dist_output: (empty) output histogram, float, shape (num_bins,)
ref_output: (empty) output codebook histogram, float, shape (num_bins,)
*/

void test_distribution
(
    at::Tensor input,
    at::Tensor dist_output,
    at::Tensor ref_output,
    float min_value,
    float max_value
)
{
    const at::cuda::OptionalCUDAGuard device_guard(input.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    TORCH_CHECK_DTYPE(input, kFloat);

    uint64_t numel = input.numel();
    uint64_t num_bins = ref_output.numel();
    TORCH_CHECK(num_bins <= MAX_BINS, "Too many bins");

    test_distribution_kernel<<<1, NUM_THREADS_TD, 0, stream>>>
    (
        (const float*) input.data_ptr(),
        (float*) dist_output.data_ptr(),
        (float*) ref_output.data_ptr(),
        numel,
        num_bins,
        min_value,
        max_value
    );
}
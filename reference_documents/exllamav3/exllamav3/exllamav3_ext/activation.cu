#include "hip/hip_runtime.h"
#include "activation.cuh"
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_fp16.h>
#include "util.h"
#include "util.cuh"
#include "compat.cuh"

#define NUM_THREADS 256
#define ACT_SILU 0
#define ACT_GELU 1

__device__ __forceinline__ half _silu(half x)
{
    half one = __float2half(1.0f);
    half neg_x = __hneg(x);
    half e = hexp(neg_x);
    half sum = __hadd(one, e);
    half r = hrcp(sum);
    half result = __hmul(x, r);
    return result;
}

__device__ __forceinline__ half2 _silu(half2 x)
{
    half2 one = __float2half2_rn(1.0f);
    half2 neg_x = __hneg2(x);
    half2 e = h2exp(neg_x);
    half2 sum = __hadd2(one, e);
    half2 r = h2rcp(sum);
    half2 result = __hmul2(x, r);
    return result;
}

__device__ __forceinline__ half _gelu(half x)
{
    float xf = __half2float(x);
    const float c = 0.797884560803f;  // sqrt(2/Pi)
    float tanh_arg = c * (xf + 0.044715f * xf * xf * xf);
    xf = 0.5f * xf * (1.0 + tanh_opt(tanh_arg));
    return __float2half_rn(xf);
}

__device__ __forceinline__ half2 _gelu(half2 x)
{
    return __halves2half2(_gelu(__low2half(x)), _gelu(__high2half(x)));
}

template <int activation_type>
__global__ __launch_bounds__(NUM_THREADS)
void act_mul_kernel
(
    const half* __restrict__ x,
    const half* __restrict__ y,
    half* __restrict__ z,
    int numel
)
{
    int idx = (blockIdx.x * NUM_THREADS + threadIdx.x);
    if (idx >= numel / 2) return;

    half2 x2 = ((const half2*) x)[idx];
    half2 y2 = ((const half2*) y)[idx];

    if constexpr (activation_type == ACT_SILU)
        x2 = _silu(x2);
    else if constexpr (activation_type == ACT_GELU)
        x2 = _gelu(x2);

    ((half2*) z)[idx] = __hmul2(x2, y2);
}

// silu(x) * y -> z, in-place if z == x or z == y

void silu_mul
(
    const at::Tensor& x,
    const at::Tensor& y,
    at::Tensor& z
)
{
    const at::cuda::OptionalCUDAGuard device_guard(x.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    int numel = x.numel();
    int blocks = CEIL_DIVIDE(numel, 2 * NUM_THREADS);
    act_mul_kernel<ACT_SILU><<<blocks, NUM_THREADS, 0, stream>>>
    (
        (const half*) x.data_ptr(),
        (const half*) y.data_ptr(),
        (half*) z.data_ptr(),
        numel
    );
}

// silu(x) * y -> z, in-place if z == x or z == y

void gelu_mul
(
    const at::Tensor& x,
    const at::Tensor& y,
    at::Tensor& z
)
{
    const at::cuda::OptionalCUDAGuard device_guard(x.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    int numel = x.numel();
    int blocks = CEIL_DIVIDE(numel, 2 * NUM_THREADS);
    act_mul_kernel<ACT_GELU><<<blocks, NUM_THREADS, 0, stream>>>
    (
        (const half*) x.data_ptr(),
        (const half*) y.data_ptr(),
        (half*) z.data_ptr(),
        numel
    );
}